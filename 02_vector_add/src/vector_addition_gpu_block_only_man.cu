
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <assert.h>

#define N 1024

__global__
void device_add( int * const __restrict__ a,
int * const __restrict__ b,
int * const __restrict__ c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__
void fill_array(int *const __restrict__ data) {
  data[blockIdx.x] = blockIdx.x;
}

__global__
void check_addition( int * const __restrict__ a,
int * const __restrict__ b,
int * const __restrict__ c) {
    assert(c[blockIdx.x] == (a[blockIdx.x]+b[blockIdx.x]));
}

int main(void) {
  int *a, *b, *c;

  int size = N * sizeof(int);

  // Alloc space for device a, b, c
  hipMallocManaged((void **)&a, size);
  hipMallocManaged((void **)&b, size);
  hipMallocManaged((void **)&c, size);

  fill_array<<<N,1>>>(a);
  fill_array<<<N,1>>>(b);

  device_add<<<N,1>>>(a,b,c);

  check_addition<<<N,1>>>(a,b,c);

  hipFree(a); hipFree(b); hipFree(c);



  return 0;
}
