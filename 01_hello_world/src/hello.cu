#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>
#include "hip/hip_runtime_api.h"

__global__ void hello(void)
{
    printf("Hello thread %d, Block: %d\n", threadIdx.x, blockIdx.x);
}

int main(void)
{

  hello<<<2,3>>>();
  getLastCudaError("hello() kernel failed");
  hipDeviceSynchronize();

  return 0;
}
