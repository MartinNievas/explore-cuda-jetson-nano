
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<string.h>
#include<omp.h>

#define NUM_THREADS 256l

#define N (1l<<20)

struct Coefficients_SOA {
  int* r;
  int* b;
  int* g;
  int* hue;
  int* saturation;
  int* maxVal;
  int* minVal;
  int* finalVal;
};


__global__
void color_conversion(Coefficients_SOA  data)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int grayscale = (data.r[i] + data.g[i] + data.b[i])/data.maxVal[i];
  int hue_sat = data.hue[i] * data.saturation[i] / data.minVal[i];

  data.finalVal[i] = grayscale*hue_sat;
}

template<typename T>
T div_round_up(T a, T b) {
  return (a + b - 1) / b;
}

int main(int argc, char*argv[])
{

  float start = 0.0, end = 0.0, elapsed = 0.0;
  Coefficients_SOA d_x;

  start = omp_get_wtime();
  hipMalloc(&d_x.r, N*sizeof(int));
  hipMalloc(&d_x.g, N*sizeof(int));
  hipMalloc(&d_x.b, N*sizeof(int));
  hipMalloc(&d_x.hue, N*sizeof(int));
  hipMalloc(&d_x.saturation, N*sizeof(int));
  hipMalloc(&d_x.maxVal, N*sizeof(int));
  hipMalloc(&d_x.minVal, N*sizeof(int));
  hipMalloc(&d_x.finalVal, N*sizeof(int));

  int num_blocks = N/NUM_THREADS;

  color_conversion<<<div_round_up(N,NUM_THREADS),NUM_THREADS>>>(d_x);

  hipFree(d_x.r);
  hipFree(d_x.g);
  hipFree(d_x.b);
  hipFree(d_x.hue);
  hipFree(d_x.saturation);
  hipFree(d_x.maxVal);
  hipFree(d_x.maxVal);
  hipFree(d_x.minVal);
  hipFree(d_x.finalVal);
  end = omp_get_wtime();
  elapsed = end-start;
  printf("Elapsed[ms]: %f\n",elapsed*1000.);

  return 0;
}


